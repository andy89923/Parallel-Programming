#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void convKernel(
    float* inp_dat, float* oup_dat, float* fil_dat,
    int imageHeight, int imageWidth, int half_fitr) {

    int poi_x = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    int poi_y = (blockIdx.y * blockDim.y + threadIdx.y);

    if (poi_x >= imageHeight || poi_y >= imageWidth) continue;

    float4 ans = (0.0, 0.0, 0.0, 0.0), tmp, fil;

    int i, j, filter_idx = 0;
    int now_x, now_y, poi;
    for (i = -half_fitr; i <= half_fitr; ++i) {

        now_x = poi_x + i;
        if (now_x < 0 || now_x >= imageHeight) continue;
        
        int wx = now_x * imageWidth;
        for (j = -half_fitr; j <= half_fitr; ++j, ++filter_idx) {
            
            now_y = poi_y + j;
            if (now_y < 0 || now_y >= imageWidth) continue;

            poi = wx + now_y;
            
            tmp = (float4)(inp_dat[poi], inp_dat[poi+1], inp_dat[poi+2], inp_dat[poi+3]);
            fil = fil_dat[filter_idx];

            ans += tmp * fil;
        }
    }
    oup_dat[gid] = ans;
}


#define BLOCK_SIZE 16

void hostFE_cuda(int filterWidth, float *filter, int imageHeight, int imageWidth,
                 float *inputImage, float *outputImage) {

    int blk_x = (imageHeight + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int blk_y = (imageWidth  + BLOCK_SIZE - 1) / BLOCK_SIZE;

    int filt_size = filterWidth * filterWidth * sizeof(float);
    int data_size = imageHeight *  imageWidth * sizeof(float);
    int half_fitr = filterWidth / 2;

    float *ans, *fit, *inp;

    hipMalloc((void**) &ans, data_size);
    hipMalloc((void**) &inp, data_size);
    hipMalloc((void**) &fit, filt_size);

    hipMemcpy(inp, inputImage, data_size, hipMemcpyHostToDevice);
    hipMemcpy(fit, filter,     filt_size, hipMemcpyHostToDevice);

    dim3 dim_block(BLOCK_SIZE, 4);
    dim3 dim_grid(blk_x, blk_y);
    convulutionKernel <<<dim_grid, dim_block>>> (inp, ans, fit, imageHeight, imageWidth, half_fitr);

    hipMemcpy(outputImage, ans, data_size, hipMemcpyDeviceToHost);

    hipFree(ans);
    hipFree(fit);
    hipFree(inp);
}